#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <xmmintrin.h>
#include <stddef.h>

__global__ void applyVel(float *wb,const float *vel,const int nxyz){
    int ix = threadIdx.x+blockIdx.x*blockDim.x;
    if(ix < nxyz)
        wb[ix]*=vel[ix];
}


int main(){
    int nx = myGrid->nx, ny = myGrid->ny, nz = myGrid->nz;
    int size=nx*ny*nz;
    size_t nBytes=size*sizeof(float);

    float *__restrict vel = volModel[VEL];

    float *wb = (float *)malloc(nBytes);

    memset(wb,0,nBytes);

    for(int i=0;i<size;i++){
        wb[i]=myLocalWavefield->wb[i];
    }

    float *d_wb;
    float *d_vel;

    hipMalloc((void **)&d_wb,nBytes);
    hipMalloc((void **)&d_vel,nBytes);

    hipMemcpy(d_vel, vel, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_wb, wb, nBytes, hipMemcpyHostToDevice);

     
    int dimx = 128;
    dim3 block(dimx);
    dim3 grid((size+block.x-1)/block.x);

    applyVel<<<grid,block>>>(d_wb,d_vel,size);

    hipMemcpy(wb,d_wb,nBytes,hipMemcpyDeviceToHost);
    for(int i=0;i<size;i++){
        myLocalWavefield->wb[i]=wb[i];
    }

    hipFree(d_wb);
    hipFree(d_vel);

    free(wb);
    free(cpu_wb);

}
