#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <xmmintrin.h>
#include <stddef.h>

__global__ void applyVel(float *wb,const float *vel,const int nxyz){
    int ix = threadIdx.x+blockIdx.x*blockDim.x;
    if(ix < nxyz)
        wb[ix]*=vel[ix];
}


int main(){
    int nx = myGrid->nx, ny = myGrid->ny, nz = myGrid->nz;
    int size=nx*ny*nz;
    size_t nBytes=size*sizeof(float);

    float *__restrict vel = volModel[VEL];

    float *mywb=myLocalWavefield->wb;

    float *d_wb;
    float *d_vel;

    hipMalloc((void **)&d_wb,nBytes);
    hipMalloc((void **)&d_vel,nBytes);

    hipMemcpy(d_vel, vel, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_wb, mywb, nBytes, hipMemcpyHostToDevice);

     
    int dimx = 128;
    dim3 block(dimx);
    dim3 grid((size+block.x-1)/block.x);

    applyVel<<<grid,block>>>(d_wb,d_vel,size);

    hipMemcpy(mywb,d_wb,nBytes,hipMemcpyDeviceToHost);

    hipFree(d_wb);
    hipFree(d_vel);

    free(wb);
    free(cpu_wb);

}
