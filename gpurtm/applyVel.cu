#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <conio.h>
#include <xmmintrin.h>
#include <stddef.h>

_global_ void applyVel(float* wb,const float *vel,const int nxyz){
    int ix = threadIdx.x+blockIdx.x*blockDim.x;
    //int iy = threadIdx.y+blockIdx.y*blockDim.y;
    //int iz = threadIdx.z+blockIdx.z*blockDim.z;
    //size_t ixz = ix * nz + iz;
    //size_t nxz = nx * nz;
    if(ix < nxyz){
        wb[ix]*=vel[i];
       // size_t i = iy * nxz+ixz;
      //  float v2 = vel[i];
       // float vectx = myLocalWavefield->wb[i];
       // myLocalWavefield->wb[i] = v2 * vectx;
    }
}



int main(){
    int nx = myGrid->nx, ny = myGrid->ny, nz = myGrid->nz;
    
    size_t size=nx*ny*nz;
    size_t nBytes=nxyz*sizeof(float);

    float *__restrict vel = volModel[VEL];

    float *wb = (float*) malloc(nBytes);
    float *h_wb = (float*) malloc(nBytes);

    for(int i=0; i<size;i++)
    {
        wb[i]=0.1f*i+0.5f;
    }

    float *d_wb;
    float *d_vel;
    hipMalloc((void **)&d_wb, nBytes);
    hipMalloc((void **)&d_vel, nBytes);

    hipMemcpy(d_vel,vel,nBytes,hipMemcpyDefault);
    hipMemcpy(d_wb,wb,nBytes,hipMemcpyDefault);
    


    int dimx = 128;
    dim3 block(dimx);
    dim3 grid((size+block.x-1)/block.x);

    applyVel<<<grid,block>>>(d_wb,d_vel,size);

    hipMemcpy(gpu_wb,d_wb,nBytes,hipMemcpyDeviceToHost);

    hipFree(d_wb);
    hipFree(d_vel);

}