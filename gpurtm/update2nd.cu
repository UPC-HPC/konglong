#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <xmmintrin.h>
#include <stddef.h>

__global__ void update2nd1(float *w0,const float *w1,const int nxyz){
    int ix = threadIdx.x+blockIdx.x*blockDim.x;
    if(ix < nxyz)
        w0[ix]=w1[ix]+w1[ix]-w0[ix];
}
__global__ void update2nd2(const float *invQ,
                           float *w0,
                           const float *w1,
                           float *d0,
                           float *d1,
                           float *cl,
                           float *el,
                           float cqsum,
                           int order,
                           const int nxyz){
    int ix = threadIdx.x+blockIdx.x*blockDim.x;

    if(ix<nxyz){
        w0[ix]=w1[ix]*(2+invQ[ix]*cqsum)-w0[ix];

        int l=0;
        for(l=0;l<order;l++)
            w0[ix]+=invQ[ix]*((1+el[l])*d0[l]-d1[l]*2);

        d0[ix]=cl[l]*w1[ix]+el[l]*d0[ix];
    }
}

int main(){
    int nx = myGrid->nx, ny = myGrid->ny, nz = myGrid->nz;
    int size = nx * ny * nz;
    size_t nBytes=size*sizeof(float);

    if(!myModel->useQ)
    {
        float *w0=(float *)malloc(nBytes);
        float *w1=(float *)malloc(nBytes);

        float *d_w0;
        float *d_w1;

        hipMalloc((void **)&d_w0,nBytes);
        hipMalloc((void **)&d_w1,nBytes);

        hipMemcpy(d_w0, w0, nBytes, hipMemcpyHostToDevice);
        hipMemcpy(d_w1, w1, nBytes, hipMemcpyHostToDevice);

        int dimx=128;
        dim3 block(dimx);
        dim3 grid((size+block.x-1)/block.x);

        update2nd1<<<grid,block>>>(d_w0,d_w1,size);
        
        hipMemcpy(w0,d_w0,nBytes,hipMemcpyDeviceToHost);

        hipFree(d_w0);
        hipFree(d_w1);

        free(w0);
        free(w1);
    }
else
    {
        float *invQ = volModel[Q];
        float *w0=(float *)malloc(nBytes);
        float *w1=(float *)malloc(nBytes);
        float cqsum=Q::cqsum;
        int   order=Q::order;
        float *d0 = myWavefield->wq[myWavefield->iq0][0];
        float *d1 = myWavefield->wq[myWavefield->iq1][0];
        float *cl = Q::cq, *el = Q::wq;

        float *d_invQ;
        float *d_w0;
        float *d_w1;
        float *d_d0;
        float *d_d1;
        float *d_cl;
        float *d_el;

        hipMalloc((void **)&d_invQ,nBytes);
        hipMalloc((void **)&d_w0,nBytes);
        hipMalloc((void **)&d_w1,nBytes);
        hipMalloc((void **)&d_d0,nBytes);
        hipMalloc((void **)&d_d1,nBytes);
        hipMalloc((void **)&d_cl,nBytes);
        hipMalloc((void **)&d_el,nBytes);
        
        hipMemcpy(d_invQ, invQ, nBytes, hipMemcpyHostToDevice);
        hipMemcpy(d_w0, w0, nBytes, hipMemcpyHostToDevice);
        hipMemcpy(d_w1, w1, nBytes, hipMemcpyHostToDevice);
        hipMemcpy(d_d0, d0, nBytes, hipMemcpyHostToDevice);
        hipMemcpy(d_d1, d1, nBytes, hipMemcpyHostToDevice);
        hipMemcpy(d_cl, cl, nBytes, hipMemcpyHostToDevice);
        hipMemcpy(d_el, el, nBytes, hipMemcpyHostToDevice);

        int dimx=128;
        dim3 block(dimx);
        dim3 grid((size+block.x-1)/block.x);

        update2nd2<<<grid,block>>>(d_invQ,d_w0,d_w1,d_d0,d_d1,d_cl,d_el,cqsum,order,size);
        
        hipMemcpy(w0,d_w0,nBytes,hipMemcpyDeviceToHost);
        hipMemcpy(d0,d_d0,nBytes,hipMemcpyDeviceToHost);

        hipFree(d_invQ);
        hipFree(d_w0);
        hipFree(d_w1);
        hipFree(d_d0);
        hipFree(d_d1);
        hipFree(d_cl);
        hipFree(d_el);
    }
}