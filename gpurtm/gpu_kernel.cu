#include "gpu_kernel.h"
#include "gpu_related.h"
inline __host__ __device__ unsigned int iDivUp(unsigned int a, unsigned int b)
{
    return (a % b != 0) ? (a / b + 1) : (a / b);
}

inline __host__ __device__ unsigned int iAlignUp(unsigned int a, unsigned int b)
{
    return (a % b != 0) ?  (a - a % b + b) : a;
}

void printGpuMemInfo () {
    size_t freeMem, totalMem;
    checkCudaErrors(hipMemGetInfo (&freeMem, &totalMem));
    printf("Free: %llu \t Total: %llu\n",
                (unsigned long long)freeMem,
                (unsigned long long)totalMem );
    return ;
}

void gpu_kernel(Wavefield *mywf)
{
    printGpuMemInfo();
    return;
}

