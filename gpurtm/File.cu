#include "hip/hip_runtime.h"
#include"hip/hip_runtime.h"
#include ""
#include<stdio.h>
using namespace std;

__global__ void divideRhoOnGPU(float* rho, float* wx, float* wy, float* wz, int nx, int ny)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(nx>1) wx[i] /= rho[i];
    if(ny>1) wy[i] /= rho[i];
    wz[i] /= rho[i];
}


void Propagator::divideRho(Wavefield* wf){ // ATTN: check the vectorization
    int nx = myGrid->nx, ny = myGrid->ny, nz = myGrid->nz;

    float* __restrict rho = volModel[RHO];
    float* __restrict wx = wf->wx;
    float* __restrict wy = wf->wy;
    float* __restrict wz = wf->wz;
    /*
#pragma omp parallel for num_threads(nThreads) schedule(static)
    for (size_t ixy = 0; ixy < nxy; ixy++) {
        for (int iz = 0; iz < nz; iz++) {
            size_t i = ixy * nz + iz;
            if (nx > 1) wx[i] /= rho[i];
            if (ny > 1) wy[i] /= rho[i];
            wz[i] /= rho[i];
        }
    }
    */

    //malloc device global memory
    float* d_rho, * d_wx, * d_wy, * d_wz;
    int nxyz = nx * ny * nz;
    int nBytes = nxyz * sizeof(float);
    
    hipMalloc((float**)&d_rho, nBytes);
    hipMalloc((float**)&d_wx, nBytes);
    hipMalloc((float**)&d_wy, nBytes);
    hipMalloc((float**)&d_wz, nBytes);

    //transfer data from host to device
    hipMemcpy(d_rho, rho, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_wx, wx, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_wy, wy, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_wz, wz, nBytes, hipMemcpyHostToDevice);

    //invoke kernel at host side
    dim3 block(128,1,1);
    dim3 grid(nxyz % 128 = 0 ? nxyz / 128 : nxyz / 128 + 1, 1, 1);

    divideRhoOnGPU << <grid, block >> > (d_rho, d_wx, d_wy, d_wz, nx, ny);
    
    //copy kernel result back to host side
    hipMemcpy(rho, d_rho, nBytes, hipMemcpyDeviceToHost);
    hipMemcpy(wx, d_wx, nBytes, hipMemcpyDeviceToHost);
    hipMemcpy(wy, d_wy, nBytes, hipMemcpyDeviceToHost);
    hipMemcpy(wz, d_wz, nBytes, hipMemcpyDeviceToHost);

    //free device global memory
    hipFree(d_rho);
    hipFree(d_wx);
    hipFree(d_wy);
    hipFree(d_wz);
}